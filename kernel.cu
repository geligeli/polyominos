#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cstdint>
#include <iostream>
#include <stdio.h>

__device__ const uint64_t *
BitmasksForTile::bitmasks_for_tile(uint64_t tile_idx) const {
  return bitmasks + bitmask_offset_for_tile[tile_idx];
}

__device__ uint64_t
BitmasksForTile::num_bitmasks_for_tile(uint64_t tile_idx) const {
  // printf("tile_idx: %d\n", tile_idx);
  return bitmask_offset_for_tile[tile_idx + 1] -
         bitmask_offset_for_tile[tile_idx];
}

__device__ void recusriveSolve(SolvingState &state, uint64_t current_state,
                               int current_index,
                               const BitmasksForTile &bitmasks) {
  // printf("current_index: %d\n", current_index);
  if (current_index == state.num_tiles) {
    state.result = true;
    return;
  }

  // printf("line: %d\n", __LINE__);

  auto tile_idx = state.candidate_tiles[current_index];

  auto *bitmask_offset = bitmasks.bitmasks_for_tile(tile_idx);
  auto num_masks = bitmasks.num_bitmasks_for_tile(tile_idx);

  std::size_t start_offset = 0;
  if (current_index > 0 && state.candidate_tiles[current_index - 1] ==
                               state.candidate_tiles[current_index]) {
    start_offset = state.indices[current_index - 1] + 1;
  }

  // printf("line: %d\n", __LINE__);
  for (std::size_t i = start_offset; i < num_masks; ++i) {
    const auto mask = bitmask_offset[i];
    if ((current_state & mask) != 0) {
      continue;
    }
    state.indices[current_index] = i;
    // printf("line: %d\n", __LINE__);

    // printf("line: %p \n", &state);
    recusriveSolve(state, current_state | mask, current_index + 1, bitmasks);
    if (state.result) {
      return;
    }
  }
  // printf("line: %d\n", __LINE__);
  state.result = false;
}

__global__ void kernel(const BitmasksForTile &bitmasks, SolvingState& state) {

  recusriveSolve(state, 0, 0, bitmasks);
}

void launch(const BitmasksForTile *device_bitmasks, SolvingState* device_state) {

  size_t newStackSize;
  CUDA_CHECK(hipDeviceGetLimit(&newStackSize, hipLimitStackSize));
  std::cout << "Old stack size: " << newStackSize << std::endl;
  CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 1024*10));
  CUDA_CHECK(hipDeviceGetLimit(&newStackSize, hipLimitStackSize));
  std::cout << "New stack size: " << newStackSize << std::endl;

  kernel<<<1, 1>>>(*device_bitmasks, *device_state);
}